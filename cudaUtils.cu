#include "hip/hip_runtime.h"
/*
 * Filename: cudaUtils.cu
 * This file is for the GPU version of the flock simulations
 */

#include "flock.h"
// void updateFlock(float * b, int NP)
// compute new positions and velocities of flocking points
// b (input/output): array of positions/velocities/accelerations
// NP (input): number of points
//
// reads velocities and accelerations
// writes positions and velocities 

__global__ void cuUpdateFlock(float * b, int NP) {
  int i;
  for (i=0; i<NP; i++) {
    pos(b, i, 0, NP) += vel(b, i, 0, NP);
    pos(b, i, 1, NP) += vel(b, i, 1, NP);

    vel(b, i, 0, NP) += acc(b, i, 0, NP);
    vel(b, i, 1, NP) += acc(b, i, 1, NP);
  }
}

// void applyNeighborForce(float *b, int NP)
//
// compute new accelerations of flocking points based on centroid
//        of neighborhood around each point
// b (input/output): array of positions/velocities/accelerations
// NP (input): number of points
//
// reads positions and velocities 
// writes accelerations

__global__ void cuApplyNeighborForce(float *b, int NP) {
  int i, j;
  for (i=0; i<NP; i++) {
    acc(b, i, 0, NP) = 0.;
    acc(b, i, 1, NP) = 0.;
    int count = 0;
    float sumX = 0., sumY = 0.;
    for (j=0; j<NP; j++) {
      if (i==j) continue;
      float sqX = pos(b, i, 0, NP) - pos(b, j, 0, NP);
      sqX *= sqX;
      float sqY = pos(b, i, 1, NP) - pos(b, j, 1, NP);
      sqY *= sqY;
      
      float diff = sqrt(sqX + sqY);
      float neighborDist = 50;
      if (diff > neighborDist) continue;
      sumX += pos(b, j, 0, NP);
      sumY += pos(b, j, 1, NP);
      count++;
    }
    if (count == 0) continue;
    sumX /= count;
    sumY /= count;

    // centroid of neighborhood is now sumX, sumY
    sumX -= pos(b, i, 0, NP);
    sumY -= pos(b, i, 1, NP);
    
    float amp = sqrt(sumX * sumX + sumY * sumY);
    float maxVel = 4;
    sumX *= maxVel / amp;
    sumY *= maxVel / amp;
    
    sumX -= vel(b, i, 0, NP);
    sumY -= vel(b, i, 1, NP);
    amp = sqrt(sumX * sumX + sumY * sumY);
    
    float maxForce = .03;
    if (amp > maxForce) {
      float f = maxForce / amp;
      sumX *= f;
      sumY *= f;
    }
    acc(b, i, 0, NP) += sumX;
    acc(b, i, 1, NP) += sumY;
  }
}


void loadBoids(FILE *fp, float *b, int NP) {
  int i, dummy, temp;
  float tx, ty;

  for (i=0; i<NP; i++) {
    temp = fscanf(fp, "%d%f%f", &dummy, &tx, &ty);
    if (temp != 3) {
      printf("Error! Input file format incorrect\n");
      return;
    }
    pos(b, i, 0, NP) = tx;
    pos(b, i, 1, NP) = ty;
  }
}

void dumpBoids(float *b, int NP) {

  FILE *fp = fopen("dump.out", "w");
  float x, y;
  int i;
  for (i=0; i<NP; i++) {
    x = pos(b, i, 0, NP);
    y = pos(b, i, 1, NP);
    fprintf(fp, "%d %f %f\n", i, x, y);
  }
  fclose(fp);
}

void dumpAccs(float *b, int NP) {

  FILE *fp = fopen("dumpAcc.out", "w");
  float x, y;
  int i;
  for (i=0; i<NP; i++) {
    x = acc(b, i, 0, NP);
    y = acc(b, i, 1, NP);
    fprintf(fp, "%d %f %f\n", i, x, y);
  }
  fclose(fp);
}
